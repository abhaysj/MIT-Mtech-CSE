#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void toggle_string(char *s, char *r, int size)
{
   int tid = threadIdx.x;
   if (tid < size)
   {
   char ch =s[tid];
   int a = (int)ch;

   int num = 0;
   while (a > 0) {
           int digit = a % 10;
           num = num * 10 + digit;
           a = a/10;
       }
    r[tid]=(char)num;

   }
}

int main()
{
    int n;
    char h_s[100],h_r[100];
    char *d_s, *d_r ;
    printf("\nEnter the string\n");
    scanf("%s",h_s);
    n = strlen(h_s);
    hipMalloc((void **)&d_s,n*sizeof(char));
    hipMalloc((void **)&d_r,n*sizeof(char));
    hipMemcpy(d_s,h_s,n*sizeof(char), hipMemcpyHostToDevice);
    toggle_string <<< 1,n>>> (d_s,d_r,n);
    hipMemcpy(h_r,d_r,n*sizeof(char), hipMemcpyDeviceToHost);
    printf("\n new reverse ACII string is \n");
      for(int i=0;i<n;i++)
       printf("%c",h_r[i]);
    hipFree(d_s);
    hipFree(d_r);
  

    return 0;


}
