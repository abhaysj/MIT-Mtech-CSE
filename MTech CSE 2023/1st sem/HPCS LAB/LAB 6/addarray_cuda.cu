#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void addArray(int *a, int *b,int *c, int size)
{
   int tid = threadIdx.x;
   if (tid < size)
   {
    c[tid] = a[tid] + b[tid];
   }
}

int main()
{
    int n;
    printf("\nEnter the size of an array:");
    scanf("%d",&n);
    int h_a[n],h_b[n],h_c[n];
    int *d_a, *d_b, *d_c;
    printf ("\n Enter array A elements");
        for (int i =0; i<n; i++)
            scanf("%d",&h_a[i]);
    printf ("\n Enter array B elements");
        for (int i =0; i<n; i++)
            scanf("%d",&h_b[i]);
    hipMalloc((void **)&d_a,n*sizeof(int));
    hipMalloc((void **)&d_b,n*sizeof(int));
    hipMalloc((void **)&d_c,n*sizeof(int));
    hipMemcpy(d_a,&h_a,n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b,&h_b,n*sizeof(int), hipMemcpyHostToDevice);
    addArray <<< 1,n>>> (d_a,d_b,d_c,n);
    hipMemcpy(&h_c,d_c,n*sizeof(int), hipMemcpyDeviceToHost);
    printf("\n sum of array elements\n");
    for(int i=0;i<n;i++)
        printf("%d ",h_c[i]);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;


}
